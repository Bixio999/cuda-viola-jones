#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <string.h>

#include "image.h"
#include "utils/common.h"

pel* readBMP_RGB(char* filename) {
	FILE* f = fopen(filename, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", filename);
		exit(1);
	}

	pel HeaderInfo[54];
	fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = abs(*(int*) &HeaderInfo[18]);
	int height = abs(*(int*) &HeaderInfo[22]);



	unsigned int i;
	//copy header for re-use
	for (i = 0; i < 54; i++)
		im.header[i] = HeaderInfo[i];

	im.height = height;
	im.width = width;
	int RowBytes = ROWBYTES(width);
	im.h_offset = RowBytes;
    im.type = "RGB";
	im.bitColor = 24;

	printf("\n   Input BMP File name: %20s  (%u x %u)", filename, im.height,
			im.width);

	pel *TheImage = (pel *) malloc(height * RowBytes * sizeof(pel));
	
	// for (i = 0; i < height * RowBytes; i++)
	// 	fread(TheImage[i], sizeof(unsigned char), 1, f);

	fread(TheImage, sizeof(unsigned char), height * RowBytes, f);

	fclose(f);
	return TheImage;  // remember to free() it in caller!
}

pel* readBMP_grey(char* filename) {
	FILE* f = fopen(filename, "rb");
	if (f == NULL) {
		printf("\n\n%s NOT FOUND\n\n", filename);
		exit(1);
	}

	pel HeaderInfo[54];
	fread(HeaderInfo, sizeof(pel), 54, f); // read the 54-byte header

	// extract image height and width from header
	int width = *(int*) &HeaderInfo[18];
	int height = *(int*) &HeaderInfo[22];

	//copy header for re-use
	unsigned int i;
	for (i = 0; i < 54; i++)
		im.header[i] = HeaderInfo[i];

	im.height = height;
	im.width = width;
	int RowBytes = width;
	im.h_offset = RowBytes;
    im.type = "GREY";
	im.bitColor = 8;

	printf("\n   Input BMP File name: %20s  (%u x %u)", filename, im.height,
			im.width);

	pel *TheImage = (pel *) malloc(height * RowBytes * sizeof(unsigned char));
	
	// for (i = 0; i < height * RowBytes; i++)
	// 	fread(TheImage + i, sizeof(unsigned char), 1, f);

	fread(TheImage, sizeof(unsigned char), height * RowBytes, f);

	fclose(f);
	return TheImage;  // remember to free() it in caller!
}

/*
 * Store a BMP image
 */
void writeBMP(pel* img, char* filename) {
	FILE* f = fopen(filename, "wb");
	if (f == NULL) {
		printf("\n\nFILE CREATION ERROR: %s\n\n", filename);
		exit(1);
	}

	unsigned int x;
	//write header
	for (x = 0; x < 54; x++)
		fputc(im.header[x], f);

	unsigned int y;
	//write data

	if (im.bitColor <= 8)
	{
		printf("\nwriting 8bit image...");
		for (x = 0; x < im.height * im.width; x++)
		{
			char temp = img[x];
			fputc(temp, f);
		}
	}
	else
	{
		printf("\nwriting 24bit image...");
		for (x = 0; x < im.height * im.h_offset; x++)
		{
			char temp = img[x];
			fputc(temp, f);
		}
	}
	

	printf("\n  Output BMP File name: %20s  (%u x %u)", filename, im.height,
			im.width);

	fclose(f);
}

__global__ void cuda_rgb2grey(pel* image, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < size)
	{
		int pixId = id * 3;

		pel grey_val = (pel) (0.3f * image[pixId + 2] + 0.59f * image[pixId + 1] + 0.11f * image[pixId]);

		image[pixId] = grey_val;
		image[pixId + 1] = grey_val;
		image[pixId + 2] = grey_val;
	}
}

pel* rgb2grey(pel* image)
{
	pel* dev_image;

	uint dimBlock = 256, dimGrid;
	int rowBlock = (im.width + dimBlock - 1) / dimBlock;
	dimGrid = im.height * rowBlock;

	printf("\n dimBlock = %d, dimGrid = %d", dimBlock, dimGrid);

	int nBytes = sizeof(pel) * im.height * im.h_offset;

	CHECK(hipMalloc((void**) &dev_image, nBytes ));
	CHECK(hipMemcpy(dev_image, image, nBytes, hipMemcpyHostToDevice));

	cuda_rgb2grey<<< dimGrid, dimBlock >>>(dev_image, im.height * im.width);

	pel* grey_image = (pel*) malloc(nBytes);

	hipDeviceSynchronize();

	CHECK(hipMemcpy(grey_image, dev_image, nBytes, hipMemcpyDeviceToHost));

	CHECK(hipFree(dev_image));

	return grey_image;
}

void write_new_BMP(char* dest_path, pel* image, int h, int w, int bitColor)
{
	FILE *f;
	unsigned char *img = NULL;
	int filesize = 54 + 3 * w * h;  //w is your image width, h is image height, both int

	img = (unsigned char *)malloc(3 * w * h);
	memset(img,0,3*w*h);

	int i, j, x, y;
	int r, g ,b;
	for(i=0; i<h; i++)
	{
		for( j=0; j<w; j++)
		{
			if (bitColor == 8)
			{
				r = image[i * h + j];
				g = r;
				b = r;
			}
			else
			{
				r = image[(i * w + j) * 3 + 2];
				g = image[(i * w + j) * 3 + 1];
				b = image[(i * w + j) * 3];
			}
			
			if (r > 255) r=255;
			if (g > 255) g=255;
			if (b > 255) b=255;

			img[( j + i * w )*3+2] = (unsigned char)(r);
			img[( j + i * w )*3+1] = (unsigned char)(g);
			img[( j + i * w )*3+0] = (unsigned char)(b);
		}
	}

	unsigned char bmpfileheader[14] = {'B','M', 0,0,0,0, 0,0, 0,0, 54,0,0,0};
	unsigned char bmpinfoheader[40] = {40,0,0,0, 0,0,0,0, 0,0,0,0, 1,0, 24,0};
	unsigned char bmppad[3] = {0,0,0};

	bmpfileheader[ 2] = (unsigned char)(filesize    );
	bmpfileheader[ 3] = (unsigned char)(filesize>> 8);
	bmpfileheader[ 4] = (unsigned char)(filesize>>16);
	bmpfileheader[ 5] = (unsigned char)(filesize>>24);

	bmpinfoheader[ 4] = (unsigned char)(       w    );
	bmpinfoheader[ 5] = (unsigned char)(       w>> 8);
	bmpinfoheader[ 6] = (unsigned char)(       w>>16);
	bmpinfoheader[ 7] = (unsigned char)(       w>>24);
	bmpinfoheader[ 8] = (unsigned char)(       h    );
	bmpinfoheader[ 9] = (unsigned char)(       h>> 8);
	bmpinfoheader[10] = (unsigned char)(       h>>16);
	bmpinfoheader[11] = (unsigned char)(       h>>24);

	f = fopen(dest_path, "wb");
	fwrite(bmpfileheader,1,14,f);
	fwrite(bmpinfoheader,1,40,f);

	for(i=0; i<h; i++)
	{
		fwrite(img+(w*(h-i-1)*3),3,w,f);
		fwrite(bmppad,1,(4-(w*3)%4)%4,f);
	}

	free(img);
	fclose(f);
}