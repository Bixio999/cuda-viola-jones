
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU (void) {
    printf("Hello World from GPU!\n");
}

int main(void) {
    // hello from GPU
    printf("Hello World from CPU!\n");
    hipSetDevice(1);
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return 0;
}




